#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "main.h"
#include ""
texture<uint2, 1, hipReadModeElementType> genoCtrl_F_Texture;
texture<uint2, 1, hipReadModeElementType> genoCtrl_M_Texture;
texture<uint2, 1, hipReadModeElementType> genoCase_F_Texture;
texture<uint2, 1, hipReadModeElementType> genoCase_M_Texture;
texture<unsigned char, 1, hipReadModeElementType> wordbits_Texture;

long long iDivUp(long long a, long long b) {
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

inline __device__ int dev_count_bit(__int64 i) {
	i = i - ((i >> 1) & 0x5555555555555555);
	i = (i & 0x3333333333333333) + ((i >> 2) & 0x3333333333333333);
	return (((i + (i >> 4)) & 0xF0F0F0F0F0F0F0F) * 0x101010101010101) >> 56;
}

inline __device__ int dev_count_bit_slow_mult(__int64 x) {
	x -= (x >> 1) & 0x5555555555555555;								//put count of each 2 bits into those 2 bits
	x = (x & 0x3333333333333333) + ((x >> 2) & 0x3333333333333333); //put count of each 4 bits into those 4 bits 
	x = (x + (x >> 4)) & 0x0f0f0f0f0f0f0f0f;						//put count of each 8 bits into those 8 bits 
	x += x >> 8;													//put count of each 16 bits into their lowest 8 bits
	x += x >> 16;													//put count of each 32 bits into their lowest 8 bits
	x += x >> 32;													//put count of each 64 bits into their lowest 8 bits
	return x & 0x7f;
}

__global__ void Screening_kernel(uint64* genoCtrl_F, uint64* genoCtrl_M, uint64* genoCase_F, uint64* genoCase_M, int nsnps, int nsamples, int nlongintCtrl_F, int nlongintCtrl_M,
	int nlongintCase_F, int nlongintCase_M, int* interactionInputOffsetJ1, int* interactionInputOffsetJ2, int *interactionPairOffsetJ1, int*interactionPairOffsetJ2,
	int* pMarginalDistrSNP, int* pMarginalDistrSNP_Y, unsigned char* wordbits)
{
	__int64 andResult = 0;
	
	int outIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int snp1 = interactionInputOffsetJ1[outIndex];
	int snp2 = interactionInputOffsetJ2[outIndex];

	int count;
	int localGenoDistr[36];
	float tao = 0;
	float InteractionMeasure = 0;
	float ptmp1, ptmp2;

	if ((snp1 >= snp2) || (snp1 >= nsnps - 1) || (snp2 >= nsnps))
	{
		return;
	}

	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			count = 0;
			for (int k = 0; k < nlongintCtrl_F; k++)
			{
				andResult = genoCtrl_F[k * 3 * nsnps + i*nsnps + snp1] & genoCtrl_F[k * 3 * nsnps + j*nsnps + snp2];
				count += dev_count_bit(andResult);
			}
			localGenoDistr[i * 3 + j] = count;
		}
	}

	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			count = 0;
			for (int k = 0; k < nlongintCase_M; k++)
			{
				andResult = genoCtrl_M[k * 3 * nsnps + i*nsnps + snp1] & genoCtrl_M[k * 3 * nsnps + j*nsnps + snp2];
				count += dev_count_bit(andResult);
			}
			localGenoDistr[9 + i * 3 + j] = count;
		}
	}

	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			count = 0;
			for (int k = 0; k < nlongintCase_F; k++)
			{
				andResult = genoCase_F[k * 3 * nsnps + i*nsnps + snp1] & genoCase_F[k * 3 * nsnps + j*nsnps + snp2];
				count += dev_count_bit(andResult);
			}
			localGenoDistr[18 + i * 3 + j] = count;
		}
	}

	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			count = 0;
			for (int k = 0; k < nlongintCase_M; k++)
			{
				andResult = genoCase_M[k * 3 * nsnps + i*nsnps + snp1] & genoCase_M[k * 3 * nsnps + j*nsnps + snp2];
				count += dev_count_bit(andResult);
			}
			localGenoDistr[27 + i * 3 + j] = count;
		}
	}

	//calculate other cells in localGenoDistr
	//control and female
	localGenoDistr[2] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 0)*nsnps + snp1] - localGenoDistr[0] - localGenoDistr[1];
	localGenoDistr[5] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 0)*nsnps + snp1] - localGenoDistr[3] - localGenoDistr[4];

	localGenoDistr[6] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 0)*nsnps + snp2] - localGenoDistr[0] - localGenoDistr[3];
	localGenoDistr[7] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 0)*nsnps + snp2] - localGenoDistr[1] - localGenoDistr[4];
	localGenoDistr[8] = pMarginalDistrSNP_Y[(2 * MarginalDistrSNP_Y_DimensionX + 0)*nsnps + snp2] - localGenoDistr[2] - localGenoDistr[5];
	
	//control and male
	localGenoDistr[11] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 1)*nsnps + snp1] - localGenoDistr[9] - localGenoDistr[10];
	localGenoDistr[14] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 1)*nsnps + snp1] - localGenoDistr[12] - localGenoDistr[13];
	
	localGenoDistr[15] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 1)*nsnps + snp2] - localGenoDistr[9] - localGenoDistr[12];
	localGenoDistr[16] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 1)*nsnps + snp2] - localGenoDistr[10] - localGenoDistr[13];
	localGenoDistr[17] = pMarginalDistrSNP_Y[(2 * MarginalDistrSNP_Y_DimensionX + 1)*nsnps + snp2] - localGenoDistr[11] - localGenoDistr[14];

	//case and female
	localGenoDistr[20] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 2)*nsnps + snp1] - localGenoDistr[18] - localGenoDistr[19];
	localGenoDistr[23] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 2)*nsnps + snp1] - localGenoDistr[21] - localGenoDistr[22];
	
	localGenoDistr[24] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 2)*nsnps + snp2] - localGenoDistr[18] - localGenoDistr[21];
	localGenoDistr[25] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 2)*nsnps + snp2] - localGenoDistr[19] - localGenoDistr[22];
	localGenoDistr[26] = pMarginalDistrSNP_Y[(2 * MarginalDistrSNP_Y_DimensionX + 2)*nsnps + snp2] - localGenoDistr[20] - localGenoDistr[23];

	//case and male
	localGenoDistr[29] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 3)*nsnps + snp1] - localGenoDistr[27] - localGenoDistr[28];
	localGenoDistr[32] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 3)*nsnps + snp1] - localGenoDistr[30] - localGenoDistr[31];

	localGenoDistr[33] = pMarginalDistrSNP_Y[(0 * MarginalDistrSNP_Y_DimensionX + 3)*nsnps + snp2] - localGenoDistr[27] - localGenoDistr[30];
	localGenoDistr[34] = pMarginalDistrSNP_Y[(1 * MarginalDistrSNP_Y_DimensionX + 3)*nsnps + snp2] - localGenoDistr[28] - localGenoDistr[31];
	localGenoDistr[35] = pMarginalDistrSNP_Y[(2 * MarginalDistrSNP_Y_DimensionX + 3)*nsnps + snp2] - localGenoDistr[29] - localGenoDistr[32];

	tao = 0;
	InteractionMeasure = 0;



}





extern "C" void cuda_GetInteractionPairs(vector<double>&InteractionMeasure, vector<pair<int, int>>& InteractionPair, uint64* genoCtrl_F, uint64* genoCtrl_M, uint64* genoCase_F, uint64* genoCase_M,
	int nsnps, int nsamples, int* nlongintCase_Gender, int* pMarginalDistrSNP, int* pMarginalDistrSNP_Y, const unsigned char* wordbits, int wordBitCount)
{
	printf("\nStarting screening ...\n");
	float timeInMs;
	hipEvent_t evStart, evStop;
	hipEventCreate(&evStart);
	hipEventCreate(&evStop);

	hipEventRecord(evStart, 0);

	uint64 *gpu_genoCtrl_F,*gpu_genoCtrl_M,*gpu_genoCase_F,*gpu_genoCase_M;
	int* gpu_pMarginalDistrSNP;
	int* gpu_pMarginalDistrSNP_Y;
	int *gpu_inputOffsetJ1;
	int *gpu_inputOffsetJ2;


	unsigned char* gpu_wordBits;
	hipMalloc((void**)&gpu_wordBits, sizeof(unsigned char)*wordBitCount);
	hipMemcpy(gpu_wordBits, wordbits, sizeof(unsigned char)*wordBitCount, hipMemcpyHostToDevice);
	hipBindTexture(0, wordbits_Texture, gpu_wordBits, sizeof(unsigned char)*wordBitCount);

	int snp1 = 0, snp2 = snp1 + 1;
	bool firstLoop = true;
	int shiftOffset = 0;
	long long totaltasks = ((long long)nsnps*(nsnps - 1)) / 2;
	long long offset = 0;

	int threadNum = THREAD_NUM;
	int blockNum = BLOCK_NUM;
	int totalNumberOfThreadBlock = iDivUp(totaltasks, (long long)threadNum);
	int totalNumberOfGridBlock = iDivUp(totalNumberOfThreadBlock, (long long)blockNum);

	int* interactionInputOffsetJ1;
	int* interactionInputOffsetJ2;
	int *gpu_InteractionPairOffsetJ1;
	int *gpu_InteractionPairOffsetJ2;
	dim3 threads(threadNum, 1, 1);
	dim3 grids(blockNum, 1, 1);

	float* gpu_floatArray;

	// normal host memory allocation
	int* interactionPairOffsetJ1 = (int *)calloc(threadNum*blockNum, sizeof(int));
	int* interactionPairOffsetJ2 = (int *)calloc(threadNum*blockNum, sizeof(int));

	
	hipHostAlloc((void**)&interactionInputOffsetJ1, sizeof(int)*blockNum*threadNum, hipHostMallocMapped);
	hipHostAlloc((void**)&interactionInputOffsetJ2, sizeof(int)*blockNum*threadNum, hipHostMallocMapped);

	//pass back the device pointer and map with host
	hipHostGetDevicePointer((void**)&gpu_inputOffsetJ1, (void*)interactionInputOffsetJ1, 0);
	hipHostGetDevicePointer((void**)&gpu_inputOffsetJ2, (void*)interactionInputOffsetJ2, 0);

	//alocate GPU memory
	hipMalloc((void**)&gpu_genoCtrl_F, sizeof(uint64)*nlongintCase_Gender[0] * 3 * nsnps);
	hipMalloc((void**)&gpu_genoCtrl_M, sizeof(uint64)*nlongintCase_Gender[1] * 3 * nsnps);
	hipMalloc((void**)&gpu_genoCase_F, sizeof(uint64)*nlongintCase_Gender[2] * 3 * nsnps);
	hipMalloc((void**)&gpu_genoCase_M, sizeof(uint64)*nlongintCase_Gender[3] * 3 * nsnps);

	hipMalloc((void**)&gpu_pMarginalDistrSNP, sizeof(int)*MarginalDistrSNP_Y_DimensionY*nsnps);
	hipMalloc((void**)&gpu_pMarginalDistrSNP_Y, sizeof(int)*MarginalDistrSNP_Y_DimensionY*MarginalDistrSNP_Y_DimensionX*nsnps);

	hipMalloc((void**)&gpu_InteractionPairOffsetJ1, sizeof(int)*threadNum*blockNum);
	hipMalloc((void**)&gpu_InteractionPairOffsetJ2, sizeof(int)*threadNum*blockNum);

	//copy geno data to GPU device and bind as texture
	hipMemcpy(gpu_genoCtrl_F, genoCtrl_F, sizeof(uint64)*nlongintCase_Gender[0] * 3 * nsnps, hipMemcpyHostToDevice);
	hipMemcpy(gpu_genoCtrl_M, genoCtrl_M, sizeof(uint64)*nlongintCase_Gender[1] * 3 * nsnps, hipMemcpyHostToDevice);
	hipMemcpy(gpu_genoCase_F, genoCase_F, sizeof(uint64)*nlongintCase_Gender[2] * 3 * nsnps, hipMemcpyHostToDevice);
	hipMemcpy(gpu_genoCase_M, genoCase_M, sizeof(uint64)*nlongintCase_Gender[3] * 3 * nsnps, hipMemcpyHostToDevice);

	hipMemcpy(gpu_pMarginalDistrSNP, pMarginalDistrSNP, sizeof(int)*MarginalDistrSNP_Y_DimensionY*nsnps,hipMemcpyHostToDevice);
	hipMemcpy(gpu_pMarginalDistrSNP_Y, pMarginalDistrSNP_Y, sizeof(int)*MarginalDistrSNP_Y_DimensionY*MarginalDistrSNP_Y_DimensionX*nsnps, hipMemcpyHostToDevice);

	hipBindTexture(0, genoCtrl_F_Texture, gpu_genoCtrl_F, sizeof(uint64)*nlongintCase_Gender[0] * 3 * nsnps);
	hipBindTexture(0, genoCtrl_M_Texture, gpu_genoCtrl_M, sizeof(uint64)*nlongintCase_Gender[1] * 3 * nsnps);
	hipBindTexture(0, genoCase_F_Texture, gpu_genoCase_F, sizeof(uint64)*nlongintCase_Gender[2] * 3 * nsnps);
	hipBindTexture(0, genoCase_M_Texture, gpu_genoCase_M, sizeof(uint64)*nlongintCase_Gender[3] * 3 * nsnps);


	for (int i = 0, offset = 0; i <= totalNumberOfGridBlock; i++, offset = offset + blockNum*threadNum)
	{
		if (i % 100 == 0)
		{
			printf("\rProgress:%d%%", (int)floor(((float)i / totalNumberOfGridBlock) * 100));
			fflush(stdout);
		}
		//snp2 = snp1 + 1;
		for (; snp1 < nsnps - 1; snp1++)
		{
			if (firstLoop)
			{
				firstLoop = false;
			}
			else
			{
				snp2 = snp1 + 1;
			}

			for (; snp2 < nsnps; snp2++)
			{
				interactionInputOffsetJ1[shiftOffset] = snp1;
				interactionInputOffsetJ2[shiftOffset] = snp2;
				shiftOffset++;

				if (shiftOffset == blockNum*threadNum)
				{
					snp2++;
					break;
				}
			}

			if (shiftOffset == blockNum*threadNum)
			{
				break;
			}
		}
		firstLoop = true;
		shiftOffset = 0;
		hipMemset(gpu_InteractionPairOffsetJ1, 0, sizeof(int)*blockNum*threadNum);
		hipMemset(gpu_InteractionPairOffsetJ2, 0, sizeof(int)*blockNum*threadNum);


	}








}